#include "hip/hip_runtime.h"
/* clang-format off */

#include "backward.cuh"
#include "bindings.h"
#include "helpers.cuh"
#include "sh.cuh"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <math.h>
#include <torch/extension.h>
#include <tuple>



namespace cg = cooperative_groups;

torch::Tensor compute_sh_backward_tensor(
    const unsigned num_points,
    const unsigned degree,
    const unsigned degrees_to_use,
    torch::Tensor &viewdirs,
    torch::Tensor &v_colors) {
    DEVICE_GUARD(viewdirs);
    if (viewdirs.ndimension() != 2 || viewdirs.size(0) != num_points ||
        viewdirs.size(1) != 3) {
        AT_ERROR("viewdirs must have dimensions (N, 3)");
    }
    if (v_colors.ndimension() != 2 || v_colors.size(0) != num_points ||
        v_colors.size(1) != 3) {
        AT_ERROR("v_colors must have dimensions (N, 3)");
    }
    unsigned num_bases = num_sh_bases(degree);
    torch::Tensor v_coeffs =
        torch::zeros({num_points, num_bases, 3}, v_colors.options());
    compute_sh_backward_kernel<<<
        (num_points + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_points,
        degree,
        degrees_to_use,
        (float3 *)viewdirs.contiguous().data_ptr<float>(),
        v_colors.contiguous().data_ptr<float>(),
        v_coeffs.contiguous().data_ptr<float>()
    );
    return v_coeffs;
}



std::tuple<
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor>
project_gaussians_backward_tensor(
    const int num_points,
    torch::Tensor &means3d,
    torch::Tensor &scales,
    const float glob_scale,
    torch::Tensor &quats,
    torch::Tensor &viewmat,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const unsigned img_height,
    const unsigned img_width,
    torch::Tensor & gaussians,
    torch::Tensor &cov3d,
    torch::Tensor &radii,
    torch::Tensor &compensation,
    torch::Tensor &v_xy,
    torch::Tensor &v_depth,
    torch::Tensor &v_conic
)
{
    DEVICE_GUARD(means3d);
    dim3 img_size_dim3;
    img_size_dim3.x = img_width;
    img_size_dim3.y = img_height;

    float4 intrins = {fx, fy, cx, cy};

    const auto num_cov3d = num_points * 6;

    // Triangular covariance.
    torch::Tensor v_cov2d =
        torch::zeros({num_points, 3}, means3d.options().dtype(torch::kFloat32));
    torch::Tensor v_cov3d =
        torch::zeros({num_points, 6}, means3d.options().dtype(torch::kFloat32));
    torch::Tensor v_mean3d =
        torch::zeros({num_points, 3}, means3d.options().dtype(torch::kFloat32));
    torch::Tensor v_scale =
        torch::zeros({num_points, 3}, means3d.options().dtype(torch::kFloat32));
    torch::Tensor v_quat =
        torch::zeros({num_points, 4}, means3d.options().dtype(torch::kFloat32));

    project_gaussians_backward_kernel<<<
        (num_points + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_points,
        (float3 *)means3d.contiguous().data_ptr<float>(),
        (float3 *)scales.contiguous().data_ptr<float>(),
        glob_scale,
        (float4 *)quats.contiguous().data_ptr<float>(),
        viewmat.contiguous().data_ptr<float>(),
        intrins,
        img_size_dim3,
        reinterpret_cast<const Gaussian*>(gaussians.data_ptr<uint8_t>()),
        cov3d.contiguous().data_ptr<float>(),
        radii.contiguous().data_ptr<int32_t>(),
        (float2 *)v_xy.contiguous().data_ptr<float>(),
        v_depth.contiguous().data_ptr<float>(),
        (float3 *)v_conic.contiguous().data_ptr<float>(),
        // Outputs.
        (float3 *)v_cov2d.contiguous().data_ptr<float>(),
        v_cov3d.contiguous().data_ptr<float>(),
        (float3 *)v_mean3d.contiguous().data_ptr<float>(),
        (float3 *)v_scale.contiguous().data_ptr<float>(),
        (float4 *)v_quat.contiguous().data_ptr<float>()
    );

    return std::make_tuple(v_cov2d, v_cov3d, v_mean3d, v_scale, v_quat);

}

std::
    tuple<
        torch::Tensor, // dL_dxy
        torch::Tensor, // dL_dconic
        torch::Tensor, // dL_dcolors
        torch::Tensor  // dL_dopacity
        >
    rasterize_backward_tensor(
        const unsigned img_height,
        const unsigned img_width,
        const unsigned block_width,
        const torch::Tensor &gaussians_ids_sorted,
        const torch::Tensor &tile_bins,
        const torch::Tensor &gaussians,
        const torch::Tensor &opacities,
        const torch::Tensor &background,
        const torch::Tensor &final_Ts,
        const torch::Tensor &final_idx,
        const torch::Tensor &v_output, // dL_dout_color
        const torch::Tensor &v_output_alpha // dL_dout_alpha
    ) {

    CHECK_INPUT(gaussians);

    const int num_points = opacities.size(0);
    const dim3 tile_bounds = {
        (img_width + block_width - 1) / block_width,
        (img_height + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {img_width, img_height, 1};
    const int channels = 3; //colors.size(1);

    torch::Tensor v_xy = torch::zeros({num_points, 2}, opacities.options());
    torch::Tensor v_conic = torch::zeros({num_points, 3}, opacities.options());
    torch::Tensor v_colors =
        torch::zeros({num_points, channels}, opacities.options());
    torch::Tensor v_opacity = torch::zeros({num_points, 1}, opacities.options());

    const Gaussian* gaussian_data_ptr = reinterpret_cast<const Gaussian*>(gaussians.data_ptr<uint8_t>());

    rasterize_backward_kernel<<<tile_bounds, block>>>(
        tile_bounds,
        img_size,
        gaussians_ids_sorted.contiguous().data_ptr<int>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        gaussian_data_ptr,
        opacities.contiguous().data_ptr<float>(),
        *(float3 *)background.contiguous().data_ptr<float>(),
        final_Ts.contiguous().data_ptr<float>(),
        final_idx.contiguous().data_ptr<int>(),
        (float3 *)v_output.contiguous().data_ptr<float>(),
        v_output_alpha.contiguous().data_ptr<float>(),
        (float2 *)v_xy.contiguous().data_ptr<float>(),
        (float3 *)v_conic.contiguous().data_ptr<float>(),
        (float3 *)v_colors.contiguous().data_ptr<float>(),
        v_opacity.contiguous().data_ptr<float>()
    );

    return std::make_tuple(v_xy, v_conic, v_colors, v_opacity);
}
